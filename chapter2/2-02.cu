#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void checkIndex(void){
    printf("threadIdx : (%d, %d, %d) blockIdx : (%d, %d, %d), blockDim : (%d, %d, %d) "
            "gridDim : (%d, %d, %d)\n", threadIdx.x , threadIdx.y, threadIdx.z, 
                                        blockIdx.x, blockIdx.y, blockIdx.z,
                                        blockDim.x, blockDim.y, blockDim.z,
                                        gridDim.x, gridDim.y, gridDim.z
            );
}

int main(int argc, char **argv){
    // データの合計数を定義
    int nElem = 6;

    // グリッドとブロックの構造を定義
    dim3 block(3);
    dim3 grid((nElem + block.x - 1) / block.x);

    // グリッドとブロックのサイズをホスト側からチェック
    printf("grid.x %d grid.y %d grid.z %d\n", grid.x, grid.y, grid.z);
    printf("block.x %d block.y %d block.z %d\n", block.x, block.y, block.z);

    // グリッドとブロックのサイズをデバイス側からもチェック
    checkIndex<<<grid, block>>> ();
    
    // デバイスリセット
    hipDeviceReset();

    return 0;
}